
#include <hip/hip_runtime.h>
#include <stdint.h>

constexpr uint32_t BLOCKDIM= 64;

__global__ void block_wide_prefix_sum_naive(float* A, uint32_t n){
    uint32_t index= threadIdx.y * BLOCKDIM  + threadIdx.x;
    
    if (index >= n){
        return;
    }

    float sum= 0.0f;
    for (uint32_t i= 0; i <= index; i++){
        sum+= A[i];
    }

    A[index]= sum;
}
extern "C" {
    void launch_block_wide_prefix_sum_naive(float* A, uint32_t n, size_t bytes){
        float* A_device;
        
        hipMalloc(&A_device, bytes);
        hipMemcpy(A_device, A, bytes, hipMemcpyHostToDevice);

        block_wide_prefix_sum_naive<<<1, BLOCKDIM>>>(A_device, n);

        hipDeviceSynchronize();

        hipMemcpy(A, A_device, bytes, hipMemcpyDeviceToHost);
        hipFree(A_device);
    }
}