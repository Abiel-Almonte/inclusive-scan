#include "hip/hip_runtime.h"
#include <stdint.h>
#include "config.h"

__device__ float block_status[MAXBLOCKS];
__device__ __forceinline__ float ks_warp_scan(float parent, uint32_t mask, uint32_t lane, uint32_t active_lanes);
__device__ __forceinline__ float make_warp_scan_exclusive(float inclusive_sum, uint32_t mask, uint32_t lane);
__device__ __forceinline__ void blelloch_cross_warp_upsweep(uint32_t tid, uint32_t num_warps, float warp_sums[]);
__device__ __forceinline__ void blelloch_cross_warp_downsweep(uint32_t tid, uint32_t num_warps, float warp_sums[]);

/*
 * Single pass device wide parallel prefix sum
 * 
 * Kogge-Stone warp scan
 * Blelloch block scan,
 * Merrill-Garland decoupled lookback.
 */
extern "C" __global__ void single_pass_scan(float* A, float* B, uint32_t N){
    
    uint32_t tid = threadIdx.x;
    uint32_t gid = blockIdx.x * blockDim.x + tid;
    
    if (tid == 0) {
        block_status[blockIdx.x] = NAN; // invalid
    }
    __syncthreads();
    
    // Local Scan
    uint32_t wid = tid / WARPSIZE;
    uint32_t lane = tid % WARPSIZE;
    uint32_t num_warps = (blockDim.x + WARPSIZE - 1) / WARPSIZE;
    
    uint32_t remaining = N - blockIdx.x * blockDim.x;
    uint32_t active_lanes = (remaining > wid * WARPSIZE) ? ((remaining >= WARPSIZE)? WARPSIZE :  remaining - wid * WARPSIZE) : 0;
    
    uint32_t mask = (active_lanes == 32) ? FULLMASK : 
        (active_lanes > 0) ? (1u << active_lanes) - 1u : 0;
    
    float x = (gid < N && active_lanes > 0) ? A[gid] : 0.0f;
    float original_x = x;
    
    if (active_lanes > 0) {
        x = ks_warp_scan(x, mask, lane, active_lanes);
    }
    
    __shared__ float warp_sums[32];
    if (tid < num_warps) {
        warp_sums[tid] = 0.0f;
    }
    __syncthreads();
    
    if (active_lanes > 0 && lane == active_lanes - 1) {
        warp_sums[wid] = x;
    }
    __syncthreads();

    blelloch_cross_warp_upsweep(tid, num_warps, warp_sums);

    if (tid == 0){
        warp_sums[num_warps - 1] = 0.0f;
    }
    __syncthreads();

    blelloch_cross_warp_downsweep(tid, num_warps, warp_sums);

    if (active_lanes > 0) {
        x = make_warp_scan_exclusive(x, mask, lane);
        if (wid > 0) {
            x += warp_sums[wid];
        }
    }

    // Global Scan

    __shared__ float block_prefix_sum;
    if(tid == blockDim.x -1){
        block_prefix_sum= x + original_x;

        if (blockIdx.x == 0){
            block_status[0] = block_prefix_sum;
        } else {
            block_status[blockIdx.x] = -block_prefix_sum; // aggregate 
        }
    }
    __syncthreads();

    // lookback
    float block_offset = 0.0f;

    if (tid == 0 && blockIdx.x > 0){
        float accumulation = 0.0f;
        for(int prev = blockIdx.x - 1; prev >= 0; prev--){
            while(isnan(block_status[prev])){/* wait */}
            if (block_status[prev] < 0.0f){
                accumulation -= block_status[prev];
            } else {
                block_offset = block_status[prev] + accumulation;
                break;
            }
        }
        block_status[blockIdx.x] = block_offset + block_prefix_sum; // prefix
        __threadfence();
    }

    __shared__ float shared_block_offset;
    if (tid == 0) {
        shared_block_offset = block_offset;
    }
    __syncthreads();

    if (gid < N && active_lanes > 0){
        B[gid] = x + shared_block_offset;
    }
}


__device__ __forceinline__ float ks_warp_scan(float parent, uint32_t mask, uint32_t lane, uint32_t active_lanes){
    for(int delta = 1; delta < active_lanes; delta <<= 1){
        float child = __shfl_up_sync(mask, parent, delta);
        if(lane >= delta && lane < active_lanes){
            parent += child;
        }
    }
    return parent;
}

__device__ __forceinline__ float make_warp_scan_exclusive(float inclusive_sum, uint32_t mask, uint32_t lane){
    float exclusive = __shfl_up_sync(mask, inclusive_sum, 1);
    if (lane == 0) {
        exclusive = 0.0f;
    }
    return exclusive;
}

__device__ __forceinline__ void blelloch_cross_warp_upsweep(uint32_t tid, uint32_t num_warps, float warp_sums[]){
    for(int delta = 1; delta <= num_warps/2; delta <<= 1){
        if (tid < num_warps) {
            uint32_t tree_index = tid + 1;
            uint32_t parent = tree_index * 2 * delta - 1;
            if (parent < num_warps){
                warp_sums[parent] += warp_sums[parent - delta];
            }
        }
        __syncthreads();
    }
}

__device__ __forceinline__ void blelloch_cross_warp_downsweep(uint32_t tid, uint32_t num_warps, float warp_sums[]){
    for(int delta = num_warps >> 1; delta >= 1; delta >>= 1){
        if (tid < num_warps) {
            uint32_t tree_index = tid + 1;
            uint32_t parent = tree_index * 2 * delta - 1;
            if (parent < num_warps){
                uint32_t child = parent - delta;
                float temp = warp_sums[parent];
                warp_sums[parent] += warp_sums[child];
                warp_sums[child] = temp;
            }
        }
        __syncthreads();
    }
}
