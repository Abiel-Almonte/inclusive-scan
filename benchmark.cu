#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <iomanip>
#include <iostream>
#include <numeric>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#include KERNEL_HEADER_PATH

#define VEC_SIZE 4

void checkCudaErrors(hipError_t err) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
        exit(-1);
    }
}

bool validate(const std::vector<float>& our_result, const std::vector<float>& cub_result, int n, float& max_rel_err) {
    max_rel_err = 0.0f;
    bool passed = true;
    for (int i = 0; i < n; ++i) {
        float ref = cub_result[i];
        float res = our_result[i];
        float rel_err = (ref != 0) ? fabsf((res - ref) / ref) : fabsf(res - ref);
        if (rel_err > 1e-5f) {
            max_rel_err = rel_err;
            passed = false;
        }
    }
    return passed;
}

int main(int argc, char** argv) {
    int N;
    if (argc > 1) {
        N = atoi(argv[1]);
    } else {
        N = 1 << 28;
    }

    std::cout << "Benchmarking for N = " << N << std::endl;

    const int N_vec = (N + VEC_SIZE - 1) / VEC_SIZE;
    const size_t n_bytes = (size_t) N * sizeof(float);

    std::vector<float> h_A(N);
    for (int i = 0; i < N; ++i)
        h_A[i] = (rand() % 100) / 100.0f;

    float *d_A, *d_B;
    checkCudaErrors(hipMalloc((void**) &d_A, n_bytes));
    checkCudaErrors(hipMalloc((void**) &d_B, n_bytes));
    checkCudaErrors(hipMemcpy(d_A, h_A.data(), n_bytes, hipMemcpyHostToDevice));

    uint32_t gridDim = (N_vec + BLOCKDIM - 1) / BLOCKDIM;
    unsigned long long* d_temp_storage_mine;
    checkCudaErrors(hipMalloc((void**) &d_temp_storage_mine, gridDim * sizeof(unsigned long long)));
    checkCudaErrors(hipMemset(d_temp_storage_mine, 0, gridDim * sizeof(unsigned long long)));

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // Warm-up run
    single_pass_scan_4x<<<gridDim, BLOCKDIM>>>((const float4*) d_A, (float4*) d_B, d_temp_storage_mine, N_vec);

    std::vector<float> timings;
    for (int i = 0; i < 101; ++i) {
        checkCudaErrors(hipEventRecord(start));
        single_pass_scan_4x<<<gridDim, BLOCKDIM>>>((const float4*) d_A, (float4*) d_B, d_temp_storage_mine, N_vec);
        checkCudaErrors(hipEventRecord(stop));
        checkCudaErrors(hipEventSynchronize(stop));
        float ms;
        checkCudaErrors(hipEventElapsedTime(&ms, start, stop));
        timings.push_back(ms);
    }
    std::sort(timings.begin(), timings.end());
    double median_ms = timings[timings.size() / 2];

    double bandwidth = (double) n_bytes * 2 / (median_ms / 1000.0) / 1e9;
    std::cout << "My Kernel Performance: " << bandwidth << " GB/s" << std::endl;

    std::vector<float> h_B(N);
    std::vector<float> h_B_cub(N);
    checkCudaErrors(hipMemcpy(h_B.data(), d_B, n_bytes, hipMemcpyDeviceToHost));

    void* d_temp_storage_cub = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(d_temp_storage_cub, temp_storage_bytes, d_A, d_B, N);
    checkCudaErrors(hipMalloc(&d_temp_storage_cub, temp_storage_bytes));

    hipcub::DeviceScan::InclusiveSum(d_temp_storage_cub, temp_storage_bytes, d_A, d_B, N);

    timings.clear();
    for (int i = 0; i < 101; ++i) {
        checkCudaErrors(hipEventRecord(start));
        hipcub::DeviceScan::InclusiveSum(d_temp_storage_cub, temp_storage_bytes, d_A, d_B, N);
        checkCudaErrors(hipEventRecord(stop));
        checkCudaErrors(hipEventSynchronize(stop));
        float ms;
        checkCudaErrors(hipEventElapsedTime(&ms, start, stop));
        timings.push_back(ms);
    }
    std::sort(timings.begin(), timings.end());
    median_ms = timings[timings.size() / 2];

    double cub_bandwidth = (double) n_bytes * 2 / (median_ms / 1000.0) / 1e9;
    checkCudaErrors(hipMemcpy(h_B_cub.data(), d_B, n_bytes, hipMemcpyDeviceToHost));
    std::cout << "CUB Kernel Performance: " << cub_bandwidth << " GB/s" << std::endl;

    float max_rel_err;
    bool passed = validate(h_B, h_B_cub, N, max_rel_err);
    if (passed) {
        std::cout << "Validation: [PASS]" << std::endl;
    } else {
        std::cout << "Validation: [FAIL] (Max Rel Err: " << std::scientific << max_rel_err << ")" << std::endl;
    }

    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_temp_storage_mine);
    hipFree(d_temp_storage_cub);

    return 0;
}