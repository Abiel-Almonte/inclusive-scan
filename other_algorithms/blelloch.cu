
#include <hip/hip_runtime.h>
#include <stdint.h>

__global__ void blelloch_scan(float* arr, float* out, uint32_t n){
    uint32_t tid = threadIdx.x;
    uint32_t gid = blockIdx.x * blockDim.x + tid;
    uint32_t tree_index = tid + 1; //assuming zero indexed array

    extern __shared__ float scratch[];
    scratch[tid] = (gid < n)? arr[gid] : 0.0f;

    uint32_t s_max = blockDim.x;

    __syncthreads();

    for (int32_t s = 1;  s <= s_max; s*=2){
        uint32_t parent = tree_index*2*s - 1;
        if (parent >= blockDim.x){
            continue;
        };

        uint32_t left_child = parent - s;

        scratch[parent] += scratch[left_child];
        __syncthreads();
    }

    if (tid == 0){
        scratch[blockDim.x -1] = 0.0f;
    }

    __syncthreads();


    for (int32_t s = s_max ;  s >= 1 ; s/=2){
        uint32_t parent = tree_index*2*s - 1;
        if (parent >= blockDim.x){
            continue;
        }
        uint32_t left_child =parent - s;

        float temp = scratch[parent];
        scratch[parent] += scratch[left_child];
        scratch[left_child] = temp;
        __syncthreads();
    }

    if (gid < n){
        out[gid] = scratch[tid];
    }
}